#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "includes/bmp_util.h"
#include "includes/kernel.cuh"
#include "includes/cpumatch.h"

int main(int argc, char *argv[]) {
  // Just an example here - you are free to modify them
  int I_width, I_height, T_width, T_height;
  float *I, *T;
  int x1, y1, x2, y2;

  // set the file location of I, T, and Output

  if (argc != 4) {
    printf("Usage: template-matching original.bmp template.bmp out.bmp\n");
    exit(0);
  }

  I = ReadBMP(argv[1], &I_width, &I_height);
  T = ReadBMP(argv[2], &T_width, &T_height);

  if (I == 0 || T == 0) {
    exit(1);
  }

  if (I_width < T_width || I_height < T_height) {
    fprintf(stderr, "Error: The template is larger than the picture\n");
    exit(EXIT_FAILURE);
  }

  int x, y;

  GetMatch(I, T, I_width, I_height, T_width, T_height, &x, &y);
  CPUGetMatch(I, T, I_width, I_height, T_width, T_height, &x, &y);
  x1 = x;
  x2 = x + T_width - 1;
  y1 = y;
  y2 = y + T_height - 1;

  MarkAndSave(argv[1], x1, y1, x2, y2, argv[3]);

  free(I);
  free(T);
  return 0;
}
