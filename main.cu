#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "includes/bmp_util.h"
#include "includes/kernel.cuh"

int main() {
  // Just an example here - you are free to modify them
  int I_width, I_height, T_width, T_height;
  float *I, *T;
  int x1, y1, x2, y2;

  // set the file location of I, T, and Output
  char I_path[] = "lena.bmp";
  char T_path[] = "lena_t.bmp";
  char out_path[] = "output.bmp";

  I = ReadBMP(I_path, &I_width, &I_height);
  T = ReadBMP(T_path, &T_width, &T_height);

  if (I_width < T_width || I_height < T_height){
    printf("the template is larger than the picture");
    return 0;
  }
  if (I == 0 || T == 0) {
    exit(1);
  }

  int x, y;

  GetMatch(I, T, I_width, I_height, T_width, T_height, &x, &y);
  x1 = x;
  x2 = x + T_width - 1;
  y1 = y;
  y2 = y + T_height - 1;

  MarkAndSave(I_path, x1, y1, x2, y2, out_path);

  free(I);
  free(T);
  return 0;
}
