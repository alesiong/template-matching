#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "includes/kernel.cuh"

__global__ void calcL1RowCumSum(const float *image, float *rowCumSum,
                                int colNumberM) {
  float sum = 0;
  for (int i = 0; i < colNumberM; ++i) {
    sum += image[threadIdx.x * colNumberM + i];
    rowCumSum[threadIdx.x * colNumberM + i] = sum;
  }
}

__global__ void calcL2RowCumSqrSum(const float *image, float *rowCumSum,
                                   int colNumberM) {
  float sum = 0;
  for (int i = 0; i < colNumberM; ++i) {
    sum += powf(image[threadIdx.x * colNumberM + i], 2);
    rowCumSum[threadIdx.x * colNumberM + i] = sum;
  }
}

__global__ void calcLxRowCumGradntSum(const float *image, float *rowCumSum,
                                      int colNumberM) {
  float sum = 0;
  for (size_t i = 0; i < colNumberM; i++) {
    sum += threadIdx.x * image[threadIdx.x * colNumberM + i];
    rowCumSum[threadIdx.x * colNumberM + i] = sum;
  }
}

__global__ void calcLyRowCumGradntSum(const float *image, float *rowCumSum,
                                      int colNumberM) {
  float sum = 0;
  for (size_t i = 0; i < colNumberM; i++) {
    sum += i * image[threadIdx.x * colNumberM + i];
    rowCumSum[threadIdx.x * colNumberM + i] = sum;
  }
}

__global__ void calcSumTable(const float *rowCumSum, float *SumTable,
                             int rowNumberN, int colNumberM) {
  for (int i = 1; i < rowNumberN; i++) {
    SumTable[i * colNumberM + blockIdx.x] +=
        rowCumSum[(i - 1) * colNumberM + blockIdx.x];
  }
}

//total (M - K + 1) * (N - K + 1) threads
//rowNumber is (N - K + 1), colNumberM is (M - K + 1)
__global__ void calcVectorFeatures(float *vectorFeatures, int rowNumberN,
                                   int colNumberM, float *l1SumTable,
                                   float *l2SumTable, float *lxSumTable,
                                   float *lySumTable, int K) {
  int start = threadIdx.x;
  float S1D = l1SumTable[start + K * colNumberM + K] - l1SumTable[start + K] \
              - l1SumTable[start + k * colNumberM] + l1SumTable[start];

  vectorFeatures[threadIdx.x * 4 + 0] = S1D / pow(K, 2);

  vectorFeatures[threadIdx.x * 4 + 1] = (l2SumTable[start + K * colNumberM + K] \
                                         - l2SumTable[start + K] \
                                         - l2SumTable[start + k * colNumberM] \
                                         + l2SumTable[start]) / pow(K, 2) \
                                  - pow(vectorFeatures[threadIdx.x * 4 + 0], 2);
  SxD = lxSumTable[start + K * colNumberM + K] - lxSumTable[start + K] \
        - lxSumTable[start + k * colNumberM] + lxSumTable[start];
  vectorFeatures[threadIdx.x * 4 + 2] = 4 * (SxD - (start / colNumberM + float(K/2))
                                        * S1D) / pow(K, 3);

  SyD = lySumTable[start + K * colNumberM + K] - lySumTable[start + K] \
        - lySumTable[start + k * colNumberM] + lySumTable[start];
  vectorFeatures[threadIdx.x * 4 + 3] = 4 * (SyD - (start % colNumberM + float(K/2))
                                        * S1D) / pow(K, 3);

}

void allocateCudaMem(float **pointer, int size) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  err = hipMalloc((void **)pointer, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device memory (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void Preprocess(const float *I, const float *T, int M, int N, int K) {
  float *l1SumTable;
  float *l2SumTable;
  float *lxSumTable;
  float *lySumTable;

  allocateCudaMem(&l1SumTable, M * N);
  allocateCudaMem(&l2SumTable, M * N);
  allocateCudaMem(&lxSumTable, M * N);
  allocateCudaMem(&lySumTable, M * N);

  float *dev_I;
  float *dev_T;
  // TODO: copy I and T to device

  hipStream_t l1Stream, l2Stream, lxStream, lyStream;
  hipStreamCreate(&l1Stream);
  hipStreamCreate(&l2Stream);
  hipStreamCreate(&lxStream);
  hipStreamCreate(&lyStream);

  // calculate l1 sum table
  calcL1RowCumSum<<<1, N, 0, l1Stream>>>(I, l1SumTable, M);
  calcL2RowCumSqrSum<<<1, N, 0, l2Stream>>>(I, l2SumTable, M);
  calcLxRowCumGradntSum<<<1, N, 0, lxStream>>>(I, lxSumTable, M);
  calcLyRowCumGradntSum<<<1, N, 0, lyStream>>>(I, lySumTable, M);

  calcSumTable<<<1, M>>>(l1SumTable, l1SumTable, N, M);

  hipStreamDestroy(l1Stream);
  hipStreamDestroy(l2Stream);
  hipStreamDestroy(lxStream);
  hipStreamDestroy(lyStream);
}

void GetMatch(float *I, float *T, int Iw, int Ih, int Tw, int Th, int *x,
              int *y) {
  Preprocess(I, T, Iw, Ih, 0);
  *x = 100;
  *y = 100;
}
